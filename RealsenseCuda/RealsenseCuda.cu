#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <librealsense2/rs.hpp>
#include <stdio.h>
#include <string>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <opencv2/opencv.hpp>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void gammaKernel(char* _dst, const char* _src, const unsigned short* _depth, int _w, float scale)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y;
	int pos = y * _w + x;

	if (x < _w)
	{
		if (_depth[pos / 3] * scale < 1.0) {
			_dst[pos] = _src[pos];
		}
		else {
			_dst[pos] = 0;
		}
	}
}

#define MAX_THREADS 128

enum class direction
{
	to_depth,
	to_color
};

int main(int argc, char* argv[]) try
{
	printf("Hello\n");
	if (!glfwInit())
	{
		printf("Failed to initialize GLFW\n");
		return -1;
	}

	int winWidth = 1280;
	int winHeight = 720;

	glfwWindowHint(GLFW_SAMPLES, 4); // 4x antialiasing
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3); // We want OpenGL 3.3
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE); // To make MacOS happy; should not be needed
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE); // We don't want the old OpenGL 

	GLFWwindow* window; // (In the accompanying source code, this variable is global for simplicity)
	window = glfwCreateWindow(winWidth, winHeight, "Tutorial 01", NULL, NULL);
	if (window == NULL) {
		fprintf(stderr, "Failed to open GLFW window. If you have an Intel GPU, they are not 3.3 compatible. Try the 2.1 version of the tutorials.\n");
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window); // Initialize GLEW
	glewExperimental = true; // Needed in core profile
	if (glewInit() != GLEW_OK) {
		fprintf(stderr, "Failed to initialize GLEW\n");
		return -1;
	}

	// Ensure we can capture the escape key being pressed below
	glfwSetInputMode(window, GLFW_STICKY_KEYS, GL_TRUE);

	GLuint to_id = 0;
	glGenTextures(1, &to_id);
	glBindTexture(GL_TEXTURE_2D, to_id);
	glTexStorage2D(GL_TEXTURE_2D, 2, GL_RGB8, 1280, 720);

	GLenum errCode1;
	if ((errCode1 = glGetError()) != GL_NO_ERROR)
	{
		printf("First: %s \n", gluErrorString(errCode1));
	}

	GLuint readFboId = 0;
	glGenFramebuffers(1, &readFboId);
	glBindFramebuffer(GL_READ_FRAMEBUFFER, readFboId);
	glFramebufferTexture2D(GL_READ_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, to_id, 0);
	glBindFramebuffer(GL_READ_FRAMEBUFFER, 0);

	GLenum errCode2;
	if ((errCode2 = glGetError()) != GL_NO_ERROR)
	{
		printf("Second: %s \n", gluErrorString(errCode2));
	}

	std::string serial;
	// Create a pipeline to easily configure and start the camera
	rs2::pipeline pipe;
	rs2::config cfg;
	if (!serial.empty())
		cfg.enable_device(serial);
	cfg.enable_stream(RS2_STREAM_COLOR, -1, 1280, 720, rs2_format::RS2_FORMAT_RGB8, 0);
	cfg.enable_stream(RS2_STREAM_DEPTH, -1, 1280, 720, rs2_format::RS2_FORMAT_Z16, 0);
	rs2::pipeline_profile profile = pipe.start(cfg);

	rs2::device dev = profile.get_device();
	rs2::depth_sensor ds = dev.query_sensors().front().as<rs2::depth_sensor>();
	float scale = ds.get_depth_scale();

	// Define two align objects. One will be used to align
	// to depth viewport and the other to color.
	// Creating align object is an expensive operation
	// that should not be performed in the main loop
	rs2::align align_to_depth(RS2_STREAM_DEPTH);
	rs2::align align_to_color(RS2_STREAM_COLOR);

	float       alpha = 0.5f;               // Transparancy coefficient
	direction   dir = direction::to_depth;  // Alignment direction

	while (glfwGetKey(window, GLFW_KEY_ESCAPE) != GLFW_PRESS && glfwWindowShouldClose(window) == 0) // Application still alive?
	{
		rs2::frameset frameset = pipe.wait_for_frames();

		if (dir == direction::to_depth){
			frameset = align_to_depth.process(frameset);
		} else {
			frameset = align_to_color.process(frameset);
		}

		auto depth = frameset.get_depth_frame();
		auto color = frameset.get_color_frame();

		const int w = color.get_width();
		const int h = color.get_height();

		int nPix = w * h;
		char* gpuImg;
		unsigned short* gpuDepthImg;
		char* gpuResImg;
		hipMalloc((void**)&gpuImg, nPix * 3 * sizeof(char));
		hipMalloc((void**)&gpuDepthImg, nPix * sizeof(unsigned short));
		hipMalloc((void**)&gpuResImg, nPix * 3 * sizeof(char));

		hipMemcpy(gpuImg, color.get_data(), nPix * 3 * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(gpuDepthImg, depth.get_data(), nPix * sizeof(unsigned short), hipMemcpyHostToDevice);

		dim3 threadBlock(MAX_THREADS);
		dim3 blockGrid((w * 3) / MAX_THREADS + 1, h, 1);

		gammaKernel << <blockGrid, threadBlock >> > (gpuResImg, gpuImg, gpuDepthImg, w * 3, scale);

		char* cpuImg;
		cpuImg = (char*)malloc(nPix * 3 * sizeof(char));

		hipMemcpy(cpuImg, gpuResImg, nPix * 3 * sizeof(char), hipMemcpyDeviceToHost);


		hipFree(gpuImg);
		hipFree(gpuDepthImg);
		hipFree(gpuResImg);

		cv::Mat my_mat(h, w, CV_8UC3, &cpuImg[0]);
		cv::cvtColor(my_mat, my_mat, cv::ColorConversionCodes::COLOR_BGR2RGB);

		cv::namedWindow("Image window", cv::WINDOW_AUTOSIZE);
		cv::imshow("Image window", my_mat);

		glBindTexture(GL_TEXTURE_2D, to_id);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, w, h, GL_RGB, GL_UNSIGNED_BYTE, cpuImg);
		//glBindFramebuffer(GL_READ_FRAMEBUFFER, readFboId);
		//glBlitFramebuffer(0, 0, w, h, 0, 0, winWidth, winHeight, GL_COLOR_BUFFER_BIT, GL_LINEAR);
		//glBindFramebuffer(GL_READ_FRAMEBUFFER, 0);
		glBindTexture(GL_TEXTURE_2D, 0);
		GLenum errCode;
		if ((errCode = glGetError()) != GL_NO_ERROR)
		{
			printf("%d: %s \n", errCode, gluErrorString(errCode));
			break;
		}

		free(cpuImg);

		if ((char)cv::waitKey(25) == 27)
			break;
		
	}

	return EXIT_SUCCESS;
}
catch (const rs2::error& e)
{
	std::cerr << "RealSense error calling " << e.get_failed_function() << "(" << e.get_failed_args() << "):\n    " << e.what() << std::endl;
	return EXIT_FAILURE;
}
catch (const std::exception& e)
{
	std::cerr << e.what() << std::endl;
	return EXIT_FAILURE;
}